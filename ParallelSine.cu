#include "hip/hip_runtime.h"
//
// Assignment 1: ParallelSine
// CSCI 415: Networking and Parallel Computation
// Spring 2017
// Name(s): Victoria Kyereme, Chengyao Tang 
//
// Sine implementation derived from slides here: http://15418.courses.cs.cmu.edu/spring2016/lecture/basicarch


// standard imports
#include <stdio.h>
#include <math.h>
#include <iomanip>
#include <iostream>
#include <string>
#include <sys/time.h>

// problem size (vector length) N
static const int N = 12345678;

//divide N by 1024 to get the number of threads

// Number of terms to use when approximating sine
static const int TERMS = 6;

// kernel function (CPU - Do not modify)
void sine_serial(float *input, float *output)
{
  int i;

  for (i=0; i<N; i++) {
      float value = input[i]; 
      float numer = input[i] * input[i] * input[i]; 
      int denom = 6; // 3! 
      int sign = -1; 
      for (int j=1; j<=TERMS;j++) 
      { 
         value += sign * numer / denom; 
         numer *= input[i] * input[i]; 
         denom *= (2*j+2) * (2*j+3); 
         sign *= -1; 
      } 
      output[i] = value; 
    }
}


// kernel function (CUDA device)
// TODO: Implement your graphics kernel here. See assignment instructions for method information

__global__ void sine_parallel(float *d_input, float *d_output)
{
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  float value = d_input[idx];
  float numer = d_input[idx] * d_input[idx] * d_input[idx];
  int denom = 6;
  int sign = -1;
  for(int j = 1; j <=TERMS; j++)
     {
        value += sign * numer / denom;
        numer *= d_input[idx] * d_input[idx];
        denom *= (2*j+2) * (2*j+3);
        sign *= -1;

     }

  d_output[idx]= value;


}

// BEGIN: timing and error checking routines (do not modify)

// Returns the current time in microseconds
long long start_timer() {
	struct timeval tv;
	gettimeofday(&tv, NULL);
	return tv.tv_sec * 1000000 + tv.tv_usec;
}


// Prints the time elapsed since the specified time
long long stop_timer(long long start_time, std::string name) {
	struct timeval tv;
	gettimeofday(&tv, NULL);
	long long end_time = tv.tv_sec * 1000000 + tv.tv_usec;
        std::cout << std::setprecision(5);	
	std::cout << name << ": " << ((float) (end_time - start_time)) / (1000 * 1000) << " sec\n";
	return end_time - start_time;
}

void checkErrors(const char label[])
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}

// END: timing and error checking routines (do not modify)



int main (int argc, char **argv)
{
  //BEGIN: CPU implementation (do not modify)
  float *h_cpu_result = (float*)malloc(N*sizeof(float));
  float *h_input = (float*)malloc(N*sizeof(float));
  //Initialize data on CPU
  int i;
  for (i=0; i<N; i++)
  {
    h_input[i] = 0.1f * i;
  }

  //Execute and time the CPU version
  long long CPU_start_time = start_timer();
  sine_serial(h_input, h_cpu_result);
  long long CPU_time = stop_timer(CPU_start_time, "\nCPU Run Time");
  //END: CPU implementation (do not modify)


  //TODO: Prepare and run your kernel, make sure to copy your results back into h_gpu_result and display your timing results

   float *h_gpu_result = (float*)malloc(N*sizeof(float));

   //Declare GPU memory pointers
   float * d_input;
   float * d_output;

   //Start time for the GPU Run Time
   long long GPU_Run_Time_start_time = start_timer();   

   //Start time for the GPU Memory Allocation
   long long GPU_Mem_Allocation_start_time = start_timer();

   //Allocate GPU memory
   hipMalloc((void**) &d_input, N*sizeof(float));
   hipMalloc((void**) &d_output, N*sizeof(float));

   //End time for the GPU Memory Allocation
   long long GPU_Mem_Allocation_end_time = stop_timer(GPU_Mem_Allocation_start_time, "\nGPU Memory Allocation");
   
   //Start time for the GPU Memory Copy to the Device
   long long GPU_Mem_Copy_Device_start_time = start_timer();
  
   //Transfer to the GPU
   hipMemcpy(d_input, h_input, N*sizeof(float), hipMemcpyHostToDevice);
   
   //End time for the GPU Memory Copy to the Device
   long long GPU_Mem_Copy_Device_end_time = stop_timer(GPU_Mem_Copy_Device_start_time, "\nGPU Memory Copy to Device");
   
   //Number of thread blocks
   int threadBlock = N/1024;

   //Start time for the GPU Kernel Run Time
   long long GPU_kernel_start_time = start_timer();

   //Launch the kernel
   sine_parallel<<<threadBlock, 1024>>>(d_output, d_input);
   
   //End time for the GPU Kernel Run Time
   long long GPU_kernel_end_time = stop_timer(GPU_kernel_start_time, "\nGPU Kernel Run Time");

   //Start time for the GPU Memory Copy to Host
   long long GPU_Mem_Copy_Host_start_time = start_timer();

   //Copy back the result to the CPU
   hipMemcpy(h_gpu_result, d_output, N*sizeof(float), hipMemcpyDeviceToHost);
  
   //End time for the GPU Memory Copy to Host
   long long GPU_Mem_Copy_Host_end_time = stop_timer(GPU_Mem_Copy_Host_start_time, "\nGPU Memory Copy to Host");

   //free GPU memory allocation
   hipFree(d_input);
   hipFree(d_output);

   //End time for the GPU Run Time
   long long GPU_Run_Time_end_time = stop_timer(GPU_Run_Time_start_time, "\nTotal GPU Run Time");


  // Checking to make sure the CPU and GPU results match - Do not modify
  int errorCount = 0;
  for (i=0; i<N; i++)
  {
    if (abs(h_cpu_result[i]-h_gpu_result[i]) > 1e-6)
      errorCount = errorCount + 1;
  }
  if (errorCount > 0)
    printf("Result comparison failed.\n");
  else
    printf("Result comparison passed.\n");

  // Cleaning up memory
  free(h_input);
  free(h_cpu_result);
  free(h_gpu_result);
  return 0;
}






